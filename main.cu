#include <iostream>
#include <math.h>
#include <cstdio>

#include "hip/hip_runtime.h"

/*
to be compiled via nvcc ==> nvcc main.cu -o exec

*/
// CUDA Kernel function to add the elements of two arrays on the GPU

bool CUDA_ = false;

// __device__ indicates funtion to be executed by the gpu
__global__ 
void mykernel(void) {

	printf( "hello from CUDA \n ==> saying hello from GPU \n"); // dunno why ==> works only with printf

}

__global__
void
summer_kernel(int* a )
{
	*a = *a +1;
}

__global__ 
void add_tut
(int *a, int *b, int *c) 
{
*c = *a + *b;
}
// __global__ functions must be void, are meant ot be called by device(GPU)
__global__
void
add(int N, int *x, int *y, int *add_result)
{	
	
	for (int i =0; i< N; ++i)
	{
		add_result[i] =  x[i] +  y[i];
	}
	 int i = blockIdx.x * blockDim.x + threadIdx.x;
	 if (i < N) {
	 add_result[i] = x[i] + y[i];
	}
}
/*
__global__
add_smart
{
c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}
*/


int main(void)
{
    int print_N_time_in_parallel = 12;
	mykernel<<<print_N_time_in_parallel,1>>>(); // function invoked on GPU
	

	// executing same operation on device:
	int execute_N_time_in_parallel = 12;

	int a_host;
	a_host = 0;

	int *a_device;
	//(1) alloc memory on GPU
	hipMalloc((void **)&a_device, sizeof(int));
	//(2) copy value(s) into GPU variable
	hipMemcpy(a_device, &a_host,sizeof(int),hipMemcpyHostToDevice);
	//(3) execute command
	summer_kernel <<< execute_N_time_in_parallel,1 >>>(a_device);
	// (4) copy back into original value
	int a_result;
	hipMemcpy(&a_result,a_device, sizeof(int), hipMemcpyDeviceToHost);// returns 1 but execute it 12 times in parallel



	
	std::cout << "EOC, sigle value ==> a_host = "<< a_result <<std::endl;
	/*
	summing 2 vector on device
	*/
	const int N = 10;

	// host arrays
    int x[N] = {  1,  2,  3,  4,  5 };
    int y[N] = { 10, 20, 30, 40, 50 };
    int z[N] = {0};

	// device copies
	int size_f = sizeof(int);

	int *d_x = 0;
	int *d_y = 0;
	int *d_z = 0;



	// vectors alllocation on GPU
	hipMalloc((void **)&d_x, size_f*N);
	hipMalloc((void **)&d_y, size_f*N);
	hipMalloc((void **)&d_z, size_f*N);

	// copy values, this operation maps device-values and host-values
	hipMemcpy(d_x, x, size_f*N, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size_f*N, hipMemcpyHostToDevice);

	//Launch add() kernel on GPU
	/*
	 Launch a kernel on the GPU with one thread for each element.
     2 is number of computational blocks and (N + 1) / 2 is a number of threads in a block
 	*/
	add<<<2, (N + 1) / 2>>>(N, d_x, d_y, d_z);

	/* cudaDeviceSynchronize waits for the kernel to finish, and returns
   		any errors encountered during the launch.*/
	hipDeviceSynchronize();

	hipMemcpy(z, d_z, size_f*N, hipMemcpyDeviceToHost);

	
	 printf("{1, 2, 3, 4, 5} + {10, 20, 30, 40, 50} = {%d, %d, %d, %d, %d}\n", z[0], z[1], z[2], z[3], z[4]);

	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = 22;
	b = 44;

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	add_tut<<<execute_N_time_in_parallel,1>>>(d_a, d_b, d_c);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	std::cout << "tutorial==> sum result is \n c = "<< c << std::endl;
	/* following line fails ahhahhahha 
	===>std::cout << "tutorial==> sum result is \n *d_c = "<< *d_c << std::endl; <=== */




	// Free memory
  	//delete [] d_x;
  	//delete [] d_y;
  	hipDeviceReset();
  	hipFree(d_x);
  	hipFree(d_y);
  	hipFree(d_a);
  	hipFree(d_b);
	







	return 0;
}